#include "hip/hip_runtime.h"
#include <cstdint>
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <ctime>
#include <stdio.h>

#include <stdlib.h>
#include <ctime>
#include <sstream>

#include "Log.h"
#include "Constants.h"

#if NVML
#include "nvml.h"
#endif

// #pragma comment(lib, "nvml.lib")
// #pragma comment(lib, "nvapi.lib")
// #pragma comment(lib, "nvapi64.lib")

#ifdef __INTELLISENSE__
#define __launch_bounds__(blocksize)
#endif

hipStream_t cudastream;

uint32_t *blockHeadermobj = nullptr;
uint32_t *midStatemobj = nullptr;
uint32_t *nonceOutmobj = nullptr;

hipError_t grindNonces(uint32_t *dev_nonceStart, uint64_t* dev_header, uint32_t* dev_nonceResult,
                        uint64_t* dev_hashStart, uint32_t *nonceResult, uint64_t *hashStart, const
                        uint64_t *header, int deviceIndex, int threadsPerBlock, int blockSize);


__device__ __forceinline__
uint2 ROR2(const uint2 a, const int offset)
{
	uint2 result;
#if __CUDA_ARCH__ > 300
	if (offset < 32) {
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
	}
	else /* if (offset < 64) */ {
		/* offset SHOULD BE < 64 ! */
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
		asm("shf.r.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
	}
#else
	if (!offset)
		result = a;
	else if (offset < 32) {
		result.y = ((a.y >> offset) | (a.x << (32 - offset)));
		result.x = ((a.x >> offset) | (a.y << (32 - offset)));
	}
	else if (offset == 32) {
		result.y = a.x;
		result.x = a.y;
	}
	else {
		result.y = ((a.x >> (offset - 32)) | (a.y << (64 - offset)));
		result.x = ((a.y >> (offset - 32)) | (a.x << (64 - offset)));
	}
#endif
	return result;
}
static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
#define B2B_G(v,a,b,c,d,x,y,c1,c2) { \
	v[a] = v[a] + v[b] + (x ^ c1); \
	v[d] ^= v[a]; \
	((uint2*)&v[d])[0] = ROR2(((uint2*)&v[d])[0], 60); \
	v[c] = v[c] + v[d]; \
	((uint2*)&v[b])[0] = ROR2(((uint2*)&v[b])[0] ^ ((uint2*)&v[c])[0], 43); \
	v[a] = v[a] + v[b] + (y ^ c2); \
	((uint2*)&v[d])[0] = ROR2(((uint2*)&v[d])[0] ^ ((uint2*)&v[a])[0], 5); \
	v[c] = v[c] + v[d]; \
	((uint2*)&v[b])[0] = ROR2(((uint2*)&v[b])[0] ^ ((uint2*)&v[c])[0], 18); \
	v[d] ^= ~(v[a] | v[b] | v[c]) | (~v[a] & v[b] & v[c]) | (v[a] & ~v[b] & v[c])   | (v[a] & v[b] & ~v[c]); \
    v[d] ^= (~v[a] & ~v[b] & v[c]) | (~v[a] & v[b] & ~v[c]) | (v[a] & ~v[b] & ~v[c]) | (v[a] & v[b] & v[c]); \
}


hipError_t grindNonces(uint32_t *nonceResult, uint64_t *hashStart, const uint64_t *header);
static __constant__ const int8_t sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },

	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13 , 0 },

	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};
__device__ __constant__
static const uint64_t u512[16] =
{
	0xA51B6A89D489E800ULL, 0xD35B2E0E0B723800ULL,
	0xA47B39A2AE9F9000ULL, 0x0C0EFA33E77E6488ULL,
	0x4F452FEC309911EBULL, 0x3CFCC66F74E1022CULL,
	0x4606AD364DC879DDULL, 0xBBA055B53D47C800ULL,
	0x531655D90C59EB1BULL, 0xD1A00BA6DAE5B800ULL,
	0x2FE452DA9632463EULL, 0x98A7B5496226F800ULL,
	0xBAFCD004F92CA000ULL, 0x64A39957839525E7ULL,
	0xD859E6F081AAE000ULL, 0x63D980597B560E6BULL
};

__global__ void vblakeHasher(const uint32_t *nonceStart, uint32_t *nonceOut, uint64_t *hashStartOut, uint64_t * __restrict__ headerIn)
{
	const uint64_t nonce = (blockDim.x * blockIdx.x + threadIdx.x) + nonceStart[0];
	uint64_t m[16] = { 0,0,0,0, 0,0,0,0 ,0,0,0,0, 0,0,0,0 };
	m[0] = headerIn[0];
	m[1] = headerIn[1];
	m[2] = headerIn[2];
	m[3] = headerIn[3];
	m[4] = headerIn[4];
	m[5] = headerIn[5];
	m[6] = headerIn[6];
	m[7] = headerIn[7];
	((uint32_t*)&m[0])[15] = nonce;

	uint64_t v[16] = {
		0x4bbf42c1f107ad85ull, 0x5D11A8C3B5AEB12Eull,
		0xA64AB78DC2774652ull, 0xC67595724658F253ull,
		0xB8864E79CB891E56ull, 0x12ED593E29FB41A1ull,
		0xB1DA3AB63C60BAA8ull, 0x6D20E50C1F954DEDull,
		0x4BBF42C1F006AD9Dull, 0x5D11A8C3B5AEB12Eull,
		0xA64AB78DC2774652ull, 0xC67595724658F253ull,
		0xb8864e79cb891e16ull, 0x12ED593E29FB41A1ull,
		0x4e25c549c39f4557ull, 0x6D20E50C1F954DEDull
	};

#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		B2B_G(v, 0, 4, 8, 12, m[sigma[i][1]], m[sigma[i][0]],
			u512[sigma[i][1]], u512[sigma[i][0]]);

		B2B_G(v, 1, 5, 9, 13, m[sigma[i][3]], m[sigma[i][2]],
			u512[sigma[i][3]], u512[sigma[i][2]]);

		B2B_G(v, 2, 6, 10, 14, m[sigma[i][5]], m[sigma[i][4]],
			u512[sigma[i][5]], u512[sigma[i][4]]);

		B2B_G(v, 3, 7, 11, 15, m[sigma[i][7]], m[sigma[i][6]],
			u512[sigma[i][7]], u512[sigma[i][6]]);

		B2B_G(v, 0, 5, 10, 15, m[sigma[i][9]], m[sigma[i][8]],
			u512[sigma[i][9]], u512[sigma[i][8]]);

		B2B_G(v, 1, 6, 11, 12, m[sigma[i][11]], m[sigma[i][10]],
			u512[sigma[i][11]], u512[sigma[i][10]]);

		B2B_G(v, 2, 7, 8, 13, m[sigma[i][13]], m[sigma[i][12]],
			u512[sigma[i][13]], u512[sigma[i][12]]);

		B2B_G(v, 3, 4, 9, 14, m[sigma[i][15]], m[sigma[i][14]],
			u512[sigma[i][15]], u512[sigma[i][14]]);
	}
	uint64_t h64 = 0x3C10ED058B3FE57E ^ v[0] ^ v[8] ^ v[3] ^ v[11] ^ v[6] ^ v[14];

	if ((h64 & 0x00000000FFFFFFFFu) == 0) {

		nonceOut[0] = nonce;
	}
}

#define SHARE_SUBMISSION_NO_RESPONSE_WARN_THRESHOLD 50

uint32_t lastNonceStart = 0;

// Grind Through vBlake nonces with the provided header, setting the resultant nonce and associated hash start if a high-difficulty solution is found
hipError_t grindNonces(uint32_t *dev_nonceStart, uint64_t* dev_header, uint32_t* dev_nonceResult,
                        uint64_t* dev_hashStart, uint32_t *nonceResult, uint64_t *hashStart, const
                        uint64_t *header, int deviceIndex, int threadsPerBlock, int blockSize)
{
	// Ensure that nonces don't overlap previous work
	uint32_t nonceStart = (uint64_t)lastNonceStart + (WORK_PER_THREAD * blockSize * threadsPerBlock);
	lastNonceStart = nonceStart;

	hipError_t cudaStatus;

	// Copy starting nonce to GPU
	cudaStatus = hipMemcpy(dev_nonceStart, &nonceStart, sizeof(uint32_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "hipMemcpy failed!");
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		hipError_t e = hipGetLastError();
		sprintf(outputBuffer, "Cuda Error: %s\n", hipGetErrorString(e));
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_header, header, 8 * sizeof(uint64_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "hipMemcpy failed!");
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		hipError_t e = hipGetLastError();
		sprintf(outputBuffer, "Cuda Error: %s\n", hipGetErrorString(e));
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}

	// Zero out hash and nonce result
	cudaStatus = hipMemset(dev_hashStart, 0, 1 * sizeof(uint64_t));
	cudaStatus = hipMemset(dev_nonceResult, 0, 1 * sizeof(uint32_t));
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "hipMemset failed!");
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		hipError_t e = hipGetLastError();
		sprintf(outputBuffer, "Cuda Error: %s\n", hipGetErrorString(e));
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}
	blockSize *= WORK_PER_THREAD;
	// Launch a kernel on the GPU with one thread for each element.
	vblakeHasher << < blockSize, threadsPerBlock >> >(dev_nonceStart, dev_nonceResult, dev_hashStart, dev_header);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "grindNonces launch failed: %s\n", hipGetErrorString(cudaStatus));
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "hipDeviceSynchronize returned error code %d after launching grindNonces!\n", cudaStatus);
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(nonceResult, dev_nonceResult, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "hipMemcpy failed!");
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		hipError_t e = hipGetLastError();
		sprintf(outputBuffer, "Cuda Error: %s\n", hipGetErrorString(e));
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}


	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(hashStart, dev_hashStart, 1 * sizeof(uint64_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		sprintf(outputBuffer, "hipMemcpy failed!");
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		hipError_t e = hipGetLastError();
		sprintf(outputBuffer, "Cuda Error: %s\n", hipGetErrorString(e));
		std::cerr << outputBuffer << endl;
		Log::error(outputBuffer);
		goto Error;
	}

Error:
	return cudaStatus;
}